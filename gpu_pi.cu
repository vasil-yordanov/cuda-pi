
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h> // CURAND lib header file
#define TRIALS_PER_THREAD 1000000
#define BLOCKS 256
#define THREADS 256
#define PI 3.14159265358979 // known value of pi 

__global__ void setup_kernel(hiprandState *states)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
	
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(tid, 0, 0, &states[tid]);
}

__global__ void pi_mc(unsigned long *estimate, hiprandState *states) 
{
	unsigned int tid = threadIdx.x + blockDim.x*blockIdx.x;
	
	unsigned long points_in_circle = 0;
	double x, y;

	hiprandState localState = states[tid];
	
	for(int i = 0; i < TRIALS_PER_THREAD; i++) 
	{
		x = hiprand_uniform(&localState);
		y = hiprand_uniform(&localState);

		points_in_circle += (x*x + y*y <= 1.0f);
	}
	
	states[tid] = localState;
	estimate[tid] = points_in_circle;
} 

int main(int argc, char *argv[]) 
{
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	  
	unsigned long host[BLOCKS * THREADS];
	unsigned long *dev;
	hiprandState *devStates;
	hipMalloc((void **) &dev, BLOCKS * THREADS * sizeof(unsigned long));
	hipMalloc( (void **)&devStates, BLOCKS*THREADS*sizeof(hiprandState) );

	unsigned long long points_in_circle = 0;
	unsigned long long total_points = 0;
	
	setup_kernel<<<BLOCKS, THREADS>>>(devStates);
	
	hipEventRecord(start);
	printf("   time (ms)  |  total points   |  points in 1/4 circle |       estimated pi        |          error        \n");
	printf("------------------------------------------------------------------------------------------------------------\n");

	for (int j=1; j < 10000; j++) 
	{ 
		pi_mc<<<BLOCKS, THREADS>>>(dev, devStates);
		hipMemcpy(host, dev, BLOCKS * THREADS * sizeof(unsigned long), hipMemcpyDeviceToHost);

		for(int i = 0; i < BLOCKS * THREADS; i++) 
		{
			points_in_circle += host[i];
		}

		total_points += (unsigned long long)BLOCKS * (unsigned long long)THREADS * (unsigned long long)TRIALS_PER_THREAD;
		
		long double pi = 4 * (long double) points_in_circle / (long double)total_points;
		
		long double error = pi - (long double) PI;
		
		hipEventRecord(stop);
		
		hipEventSynchronize(stop);
		
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		
		printf("%14.0f\t%16lld\t%16lld\t%20.14lf\t%20.14lf\n", milliseconds, total_points, points_in_circle, pi, error);
	}
	hipFree(dev);
	hipFree(devStates);
 return 0;
} 
